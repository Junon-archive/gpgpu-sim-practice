
#include <hip/hip_runtime.h>
#include <stdio.h>

__global__ void hello_kernel() {
    int idx = threadIdx.x + blockIdx.x * blockDim.x;
    printf("Hello from thread %d\n", idx);
}

int main() {
    // 간단한 커널 실행
    hello_kernel<<<2, 4>>>();
    hipDeviceSynchronize(); // 커널이 끝날 때까지 대기
    return 0;
}