
#include <hip/hip_runtime.h>
#include <stdio.h>
#define N 128

__global__ void matrixMul(float* A, float* B, float* C) {
    int row = threadIdx.y + blockDim.y * blockIdx.y;
    int col = threadIdx.x + blockDim.x * blockIdx.x;

    float sum = 0.0f;
    for (int k = 0; k < N; ++k)
        sum += A[row * N + k] * B[k * N + col];

    C[row * N + col] = sum;
}

int main() {
    int size = N * N * sizeof(float);
    float *A, *B, *C;
    float *d_A, *d_B, *d_C;

    A = (float*)malloc(size);
    B = (float*)malloc(size);
    C = (float*)malloc(size);

    for (int i = 0; i < N * N; ++i) {
        A[i] = 1.0f;
        B[i] = 2.0f;
    }

    hipMalloc(&d_A, size);
    hipMalloc(&d_B, size);
    hipMalloc(&d_C, size);

    hipMemcpy(d_A, A, size, hipMemcpyHostToDevice);
    hipMemcpy(d_B, B, size, hipMemcpyHostToDevice);

    dim3 threadsPerBlock(16, 16);
    dim3 numBlocks(N / threadsPerBlock.x, N / threadsPerBlock.y);
    matrixMul<<<numBlocks, threadsPerBlock>>>(d_A, d_B, d_C);

    hipMemcpy(C, d_C, size, hipMemcpyDeviceToHost);

    // 검증
    printf("C[0] = %f\n", C[0]);

    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);
    free(A); free(B); free(C);
    return 0;
}
