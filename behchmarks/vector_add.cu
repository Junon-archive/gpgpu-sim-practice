// vectorAdd.cu

#include <hip/hip_runtime.h>
#include <stdio.h>

__global__ void vectorAdd(const float *A, const float *B, float *C, int N) {
    int i = threadIdx.x + blockDim.x * blockIdx.x;
    if (i < N) C[i] = A[i] + B[i];
}

int main() {
    int N = 500;
    size_t size = N * sizeof(float);
    float *h_A = (float *)malloc(size);
    float *h_B = (float *)malloc(size);
    float *h_C = (float *)malloc(size);

    for (int i = 0; i < N; i++) {
        h_A[i] = i;
        h_B[i] = i * 2;
    }

    float *d_A, *d_B, *d_C;
    hipMalloc(&d_A, size);
    hipMalloc(&d_B, size);
    hipMalloc(&d_C, size);

    hipMemcpy(d_A, h_A, size, hipMemcpyHostToDevice);
    hipMemcpy(d_B, h_B, size, hipMemcpyHostToDevice);

    vectorAdd<<<(N+255)/256, 256>>>(d_A, d_B, d_C, N);

    hipMemcpy(h_C, d_C, size, hipMemcpyDeviceToHost);

    for (int i = 0; i < 10; i++) {
        printf("%f\n", h_C[i]);
    }

    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);
    free(h_A);
    free(h_B);
    free(h_C);

    return 0;
}
